#include "hip/hip_runtime.h"
#include "cufft_routines.cuh"

// Complex conjugate
__device__ hipfftComplex 
complex_conj(hipfftComplex a)
{
    hipfftComplex c;
    c.x =  a.x;
    c.y = -a.y;
    return c;
}

// Complex scale
__device__ hipfftComplex 
complex_scale(hipfftComplex a, float s)
{
    hipfftComplex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

// Complex multiplication
__device__ hipfftComplex
complex_mul(hipfftComplex a, hipfftComplex b)
{
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

// Complex pointwise conjugation/multiplication/scaling of arrays
// result is stored 
__global__ void 
complex_pointwise_cms(hipfftComplex* a, const hipfftComplex* b,
    int size, float scale)
{
    unsigned long i;
    hipfftComplex atb;
    hipfftComplex atbs;
    hipfftComplex bconj;

    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    for (i = threadID; i < size; i += numThreads)
    {
        bconj = complex_conj(b[i]);
        atb = complex_mul(a[i], bconj);
        atbs = complex_scale(atb, scale);
        a[i] = atbs;
    }
}

__global__ void 
copy_halfforward_halfreverse(hipfftComplex* a, hipfftComplex* out, 
    long nsamples)
{
    unsigned long i;
    unsigned long j;

    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    for(i=threadID; i < nsamples; i += numThreads)
    {
        out[i] = out[i+nsamples];
    }
    for(i=threadID, j=nsamples*2-1; i<nsamples; i+=numThreads, j-=numThreads)
    {
        out[i+nsamples] = out[j];
    }
}
