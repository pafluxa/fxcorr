#include <stdio.h>
#include "fxcorr.cuh"
#include "cufft_routines.cuh"

void _cudaCheck(hipError_t err, const char *file, int line) {
   if (err != hipSuccess) 
   {
       fprintf(stderr, "cuda error: %s at %s line %d", hipGetErrorString(err), file, line);
       exit(-1);
   }
}
#define cudaCheck(ans) { _cudaCheck((ans), __FILE__, __LINE__); }

extern void 
fxcorr_attach_signals(float* s1, float* s2, long nsamples,
    fxcorrData* fxd)
{
    fxd->nsamples = nsamples;
    fxd->signal1 = s1;
    fxd->signal2 = s2; 
    fxd->out = (float*)malloc(2*sizeof(float)*nsamples);
};

extern void
fxcorr_allocate_and_zeropad(fxcorrData* fxd) // TODO: add CUDA checks
{
    // two is there because of zero-padding
    fxd->sz = (2*fxd->nsamples)*sizeof(hipfftComplex);
    // allocate memory on device
    cudaCheck(hipMalloc((void**)&fxd->cu_signal1, fxd->sz));
    cudaCheck(hipMalloc((void**)&fxd->cu_signal2, fxd->sz));
    // memset is the fastest way to zero-pad
    cudaCheck(hipMemset(fxd->cu_signal1, 0, fxd->sz));
    cudaCheck(hipMemset(fxd->cu_signal2, 0, fxd->sz));
    // flag that buffers are good
    fxd->hasCudaBuffers = 1;
}

extern void 
fxcorr_deallocate(fxcorrData* fxd)
{
    //Destroy CUFFT context
    hipfftDestroy(fxd->cufft_plan);

    // cleanup memory
    hipFree(fxd->cu_signal1);
    hipFree(fxd->cu_signal2);
    
    free(fxd->out);
}

extern int
fxcorr_create_cufft_handle(fxcorrData* fxd)
{
    if(hipfftPlan1d(&(fxd->cufft_plan), 
        fxd->nsamples*2, HIPFFT_C2C, 1) != HIPFFT_SUCCESS )
    {
        fprintf(stderr, "CUDA FFT plan creation has failed.\n");
        return -1;
    }
    fxd->hasCUFFTPlan = 1;
    
    return 0;
}

extern int
fxcorr_compute(fxcorrData* fxd)
{
    // return -1 if no buffers are present
    if(fxd->hasCudaBuffers != 1)
    { 
        return -1;
    }
    
    // copy host data to GPU
    hipMemcpy(fxd->cu_signal1, fxd->signal1, fxd->sz,
        hipMemcpyHostToDevice);
    hipMemcpy(fxd->cu_signal2, fxd->signal2, fxd->sz,
        hipMemcpyHostToDevice);
    
    // transform signal1 *IN-PLACE*
    if(hipfftExecC2C(
        fxd->cufft_plan, 
        fxd->cu_signal1, fxd->cu_signal1, 
        HIPFFT_FORWARD) != HIPFFT_SUCCESS )
    {
        fprintf(stderr, "Launching C2C FFT has failed.\n");
        return -1;
    }
    
    // transform signal2 *IN-PLACE*
    if(hipfftExecC2C(
        fxd->cufft_plan, 
        fxd->cu_signal2, fxd->cu_signal2, 
        HIPFFT_FORWARD) != HIPFFT_SUCCESS )
    {
        fprintf(stderr, "Launching C2C FFT has failed.\n");
        return -1;
    }
    /*
    // calls above are asynchronous. explicit synchronization hurts
    // but it is needed this time.
    if(hipDeviceSynchronize() != hipSuccess)
	{
		fprintf(stderr, "CUDA error: failed to synchronize!.\n"); 
        // if synchronization fails, it might still be possible to 
        // keep going, but I rather halt everything because it is an
        // indicator that something isn't working as intended.
        return -1;
	}
    */
    // multiply the coefficients together and normalize the result
    float T = 1.0/(2*fxd->nsamples - 1);
    // 32 -> block size of 32 (which is the warp size of most SMU 
    // in a variety of GPUs. 
    // 256 -> fine tune at will
    // result of this operation is held at fxd->cu_signal1
    complex_pointwise_cms<<<32, 256>>>(
        fxd->cu_signal1, fxd->cu_signal2, 2*fxd->nsamples, T);
    // transform back
    if(hipfftExecC2C(
        fxd->cufft_plan, 
        fxd->cu_signal1, fxd->cu_signal1, 
        HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
    {
        fprintf( stderr, "Launching C2C inverse FFT has failed.\n" );
        return -1;
    }

    // copy fourier transform back to host
    hipfftComplex* correlation = (hipfftComplex*)malloc(
        sizeof(hipfftComplex) * fxd->nsamples*2 );;
    hipMemcpy(
        correlation, fxd->cu_signal1, 
        sizeof(hipfftComplex)*fxd->nsamples*2, 
        hipMemcpyDeviceToHost);

    // Copy correlation to host. First half in ascending order
    for( int i=0; i < fxd->nsamples ; i++ )
    {    
        fxd->out[i+fxd->nsamples] = correlation[fxd->nsamples+i].x/fxd->nsamples;
    }
    // Copy correlation to host. Second half in descending order
    for( int i=0, j=2*fxd->nsamples-1; i < fxd->nsamples ; i++,j-- )
    {    
        fxd->out[i] = correlation[j].x/fxd->nsamples;
    }    
    
    // free temporal buffer
    hipFree(correlation);
    
    return 0;
}
